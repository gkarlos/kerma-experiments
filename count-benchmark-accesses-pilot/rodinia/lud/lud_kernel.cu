#include <hip/hip_runtime.h>
#include <stdio.h>

#ifdef RD_WG_SIZE_0_0
        #define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE RD_WG_SIZE
#else
        #define BLOCK_SIZE 16
#endif


__global__ void lud_diagonal(float *m, int matrix_dim, int offset)
{
  int i,j;
  __shared__ float shadow[BLOCK_SIZE][BLOCK_SIZE];

  int array_offset = offset * matrix_dim + offset;

  for(i=0; i < BLOCK_SIZE; i++){
    shadow[i][threadIdx.x] = m[array_offset+threadIdx.x];
    array_offset += matrix_dim;
  }
  // 2/2/2/0

  __syncthreads();
  
  for(i=0; i < BLOCK_SIZE-1; i++) {

    if (threadIdx.x>i){
      for(j=0; j < i; j++)
        shadow[threadIdx.x][i] -= shadow[threadIdx.x][j] * shadow[j][i];
      // 6/6/6/0
      shadow[threadIdx.x][i] /= shadow[i][i];
      // 9/9/9/0
    }

    __syncthreads();

    if (threadIdx.x>i){

      for(j=0; j < i+1; j++)
        shadow[i+1][threadIdx.x] -= shadow[i+1][j] * shadow[j][threadIdx.x];
        // 13/13/13/0
    }
    __syncthreads();
  }

  /* 
     The first row is not modified, it
     is no need to write it back to the
     global memory

   */
  array_offset = (offset+1) * matrix_dim + offset;
  for(i=1; i < BLOCK_SIZE; i++){
    m[array_offset+threadIdx.x]=shadow[i][threadIdx.x];
    // 15/15/15/0
    array_offset += matrix_dim;
  }
}

__global__ void lud_perimeter(float *m, int matrix_dim, int offset)
{
  __shared__ float dia[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i,j, array_offset;
  int idx;

  if (threadIdx.x < BLOCK_SIZE) {
    idx = threadIdx.x;
    
    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE/2; i++){
      dia[i][idx]=m[array_offset+idx];
      //2/2/2/0
      array_offset += matrix_dim;
    }
    
    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      peri_row[i][idx]=m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx];
      //4/4/4/0
      array_offset += matrix_dim;
    }

  } else {
    idx = threadIdx.x-BLOCK_SIZE;
    
    array_offset = (offset+BLOCK_SIZE/2)*matrix_dim+offset;
    for (i=BLOCK_SIZE/2; i < BLOCK_SIZE; i++){
      dia[i][idx]=m[array_offset+idx];
      //6/6/6/0
      array_offset += matrix_dim;
    }
    
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      peri_col[i][idx] = m[array_offset+idx];
      // 8/8/8/0
      array_offset += matrix_dim;
    }
  
  }
  __syncthreads();

  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    for(i=1; i < BLOCK_SIZE; i++){
      for (j=0; j < i; j++)
        peri_row[i][idx]-=dia[i][j]*peri_row[j][idx];
        // 12/12/12/0
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    for(i=0; i < BLOCK_SIZE; i++){
      for(j=0; j < i; j++)
        peri_col[idx][i]-=peri_col[idx][j]*dia[j][i];
      // 16/16/16/0
      peri_col[idx][i] /= dia[i][i];
      // 19/19/19/0
    }
  }

  __syncthreads();
    
  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    array_offset = (offset+1)*matrix_dim+offset;
    for(i=1; i < BLOCK_SIZE; i++){
      m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
      array_offset += matrix_dim;
      // 21/21/21/0
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for(i=0; i < BLOCK_SIZE; i++){
      m[array_offset+idx] =  peri_col[i][idx];
      // 23/23/23/0
      array_offset += matrix_dim;
    }
  }

}

__global__ void lud_internal(float *m, int matrix_dim, int offset)
{
  __shared__ float peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i;
  float sum;

  int global_row_id = offset + (blockIdx.y+1)*BLOCK_SIZE;
  int global_col_id = offset + (blockIdx.x+1)*BLOCK_SIZE;

  peri_row[threadIdx.y][threadIdx.x] = m[(offset+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x];
  peri_col[threadIdx.y][threadIdx.x] = m[(global_row_id+threadIdx.y)*matrix_dim+offset+threadIdx.x];

  // 4/4/0/0
  __syncthreads();

  sum = 0;
  for (i=0; i < BLOCK_SIZE; i++)
    sum += peri_col[threadIdx.y][i] * peri_row[i][threadIdx.x];
  // 6/6/2/0
  m[(global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x] -= sum;
  // 8/8/2/0


}


void lud_cuda(float *m, int matrix_dim)
{
  int i=0;
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  float *m_debug = (float*)malloc(matrix_dim*matrix_dim*sizeof(float));

  for (i=0; i < matrix_dim-BLOCK_SIZE; i += BLOCK_SIZE) {
      lud_diagonal<<<1, BLOCK_SIZE>>>(m, matrix_dim, i);
      lud_perimeter<<<(matrix_dim-i)/BLOCK_SIZE-1, BLOCK_SIZE*2>>>(m, matrix_dim, i);
      dim3 dimGrid((matrix_dim-i)/BLOCK_SIZE-1, (matrix_dim-i)/BLOCK_SIZE-1);
      lud_internal<<<dimGrid, dimBlock>>>(m, matrix_dim, i); 
  }
  lud_diagonal<<<1,BLOCK_SIZE>>>(m, matrix_dim, i);
}

