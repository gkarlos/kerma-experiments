#include "hip/hip_runtime.h"
#include "srad.h"
#include <stdio.h>

__global__ void srad_cuda_1(float *E_C,
                            float *W_C,
                            float *N_C,
                            float *S_C,
                            float * J_cuda,
                            float * C_cuda,
                            int cols,
                            int rows,
                            float q0sqr
)
{

  //block id
  int bx = blockIdx.x;
  int by = blockIdx.y;

  //thread id
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  //indices
  int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
  int index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + tx - cols;
  int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
  int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
  int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;

  float n, w, e, s, jc, g2, l, num, den, qsqr, c;

  //shared memory allocation
  __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float temp_result[BLOCK_SIZE][BLOCK_SIZE];

  __shared__ float north[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float south[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float  east[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float  west[BLOCK_SIZE][BLOCK_SIZE];

  //load data to shared memory
  north[ty][tx] = J_cuda[index_n]; 
  south[ty][tx] = J_cuda[index_s];
  // 4/4/0/0

  if ( by == 0 ){
    north[ty][tx] = J_cuda[BLOCK_SIZE * bx + tx]; 
    // 6/6/0/0
  }
  else if ( by == gridDim.y - 1 ){
    south[ty][tx] = J_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
    // 8/8/0/0
  }
   __syncthreads();
 
  west[ty][tx] = J_cuda[index_w];
  east[ty][tx] = J_cuda[index_e];
  // 10/10/0/0

  if ( bx == 0 ){
    west[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + cols * ty]; 
    // 12/12/0/0
  }
  else if ( bx == gridDim.x - 1 ){
    east[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
    // 14/14/0/0
  }
 
  __syncthreads();

  temp[ty][tx] = J_cuda[index];
  // 16/16/0/0

  __syncthreads();

  jc = temp[ty][tx];
  // 17/17/0/0

  if ( ty == 0 && tx == 0 ){ //nw
    n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx]  - jc;
    e  = temp[ty][tx+1] - jc;
    // 21/21/0/0
  }
  else if ( ty == 0 && tx == BLOCK_SIZE-1 ){ //ne
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc;
    e  = east[ty][tx] - jc;
    // 25/25/0/0
  }
  else if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc;
    e  = east[ty][tx]  - jc;
    // 29/29/0/0
  }
  else if ( ty == BLOCK_SIZE -1 && tx == 0 ){//sw
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = west[ty][tx]  - jc; 
    e  = temp[ty][tx+1] - jc;
    // 33/33/0/0
  }
  else if ( ty == 0 ){ //n
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
    // 37/37/0/0
  }
  else if ( tx == BLOCK_SIZE -1 ){ //e
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx] - jc;
    // 41/41/0/0
  }
  else if ( ty == BLOCK_SIZE -1){ //s
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
    // 45/45/0/0
  }
  else if ( tx == 0 ){ //w
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx] - jc; 
    e  = temp[ty][tx+1] - jc;
    // 49/49/0/0
  }
  else{  //the data elements which are not on the borders 
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc;
    e  = temp[ty][tx+1] - jc;
    //53/53/0/0
  }


  g2 = ( n * n + s * s + w * w + e * e ) / (jc * jc);

  l = ( n + s + w + e ) / jc;

  num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
  den  = 1 + (.25*l);
  qsqr = num/(den*den);

  // diffusion coefficent (equ 33)
  den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
  c = 1.0 / (1.0+den) ;

  // saturate diffusion coefficent
  if (c < 0){
    temp_result[ty][tx] = 0;
  }
  else if (c > 1) {
    temp_result[ty][tx] = 1;
  }
  else {
    temp_result[ty][tx] = c;
  }
  //56/56/0/0

  __syncthreads();

  C_cuda[index] = temp_result[ty][tx];
  E_C[index] = e;
  W_C[index] = w;
  S_C[index] = s;
  N_C[index] = n;
  //62/62/0/0

}

__global__ void srad_cuda_2(float *E_C, 
                            float *W_C, 
                            float *N_C, 
                            float *S_C,	
                            float * J_cuda, 
                            float * C_cuda, 
                            int cols, 
                            int rows, 
                            float lambda,
                            float q0sqr
) 
{
	//block id
	int bx = blockIdx.x;
    int by = blockIdx.y;

	//thread id
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	//indices
    int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
	int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
    int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
	float cc, cn, cs, ce, cw, d_sum;

	//shared memory allocation
	__shared__ float south_c[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float  east_c[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float c_cuda_temp[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float c_cuda_result[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];

    //load data to shared memory
	temp[ty][tx] = J_cuda[index];
    // 2/2/0/0

    __syncthreads();
	 
	south_c[ty][tx] = C_cuda[index_s];
    // 4/4/0/0

	if ( by == gridDim.y - 1 ){
	  south_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
	}
    // 6/6/0/0

	__syncthreads();

	east_c[ty][tx] = C_cuda[index_e];
    // 8/8/0/0

	if ( bx == gridDim.x - 1 ){
	  east_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
	}
	//10/10/0/0
    __syncthreads();

    c_cuda_temp[ty][tx]      = C_cuda[index];
    //12/12/0/0

    __syncthreads();

	cc = c_cuda_temp[ty][tx];
    //13/13/0/0

    if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
        cn  = cc;
        cs  = south_c[ty][tx];
        cw  = cc; 
        ce  = east_c[ty][tx];
        // 15/15/0/0
    }
    else if ( tx == BLOCK_SIZE -1 ){ //e
        cn  = cc;
        cs  = c_cuda_temp[ty+1][tx];
        cw  = cc; 
        ce  = east_c[ty][tx];
        // 17/17/0/0
    }
    else if ( ty == BLOCK_SIZE -1){ //s
        cn  = cc;
        cs  = south_c[ty][tx];
        cw  = cc; 
        ce  = c_cuda_temp[ty][tx+1];
        // 19/19/0/0
    }
    else{ //the data elements which are not on the borders 
        cn  = cc;
        cs  = c_cuda_temp[ty+1][tx];
        cw  = cc; 
        ce  = c_cuda_temp[ty][tx+1];
        // 21/21/0/0
    }

   // divergence (equ 58)
   d_sum = cn * N_C[index] + cs * S_C[index] + cw * W_C[index] + ce * E_C[index];
   // 25/25/0/0

   // image update (equ 61)
   c_cuda_result[ty][tx] = temp[ty][tx] + 0.25 * lambda * d_sum;
   // 27/27/0/0

   __syncthreads();

   J_cuda[index] = c_cuda_result[ty][tx];
   // 29/29/0/0
}
